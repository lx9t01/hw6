#include "hip/hip_runtime.h"
// gillespie.cu Haixiang Xu
#include <cstdio>
#include <cmath>

#include <hip/hip_runtime.h>
#include "gillespie_cuda.cuh"

__device__ static float atomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}
/*
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}
*/


// a single iteration of the Gillespie algorithm on 
// the given system using an array of random numbers 
// given as an argument.
__global__
void cudaGillKernel(float* dev_points,
    float* dev_points_2,
    float* state,
    float* X, 
    float* dev_timestep,
    float* dev_accu_time,
    const int N) {

    const float kon = 0.1;
    const float koff = 0.9;
    const float b = 10.0;
    const float g = 1.0;

    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    while (idx < N) {

        if (state[idx] < 0.5){
            float lamda = kon + X[idx] * g;
            dev_timestep[idx] = -logf(dev_points[idx]) / lamda;
            dev_accu_time[idx] += dev_timestep[idx];
            if (dev_points_2[idx] > kon / lamda) { // if X--
                X[idx]--;
            } else { // if OFF --> ON
                state[idx] = 1;
            }
        } else {
            float lamda = koff + b + X[idx] * g;
            dev_timestep[idx] = -logf(dev_points[idx]) / lamda;
            dev_accu_time[idx] += dev_timestep[idx];
            if (dev_points_2[idx] <= koff / lamda) { // ON --> OFF
                state[idx] = 0;
            } else if (dev_points_2[idx] <= (koff + b) / lamda) { // X++
                X[idx]++;
            } else { // X--
                X[idx]--;
            }
        }
        __syncthreads();
        idx += blockDim.x * gridDim.x;
    }
}

// a kernel to use reduction to find minimum
__global__
void cudaFindMinKernel (
    float* dev_timestep,
    float* min_timestep, // NEED TO ALLOCATE
    const int N) {

    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ float data[64]; // rememeber to update this !!!!!!!

    for (int i = 0; i < 64; ++i){
        data[i] = 99999;
    }
    __syncthreads();

    while (idx < N) {
        {
            atomicMin(&data[threadIdx.x], dev_timestep[idx]);
        }
        idx += blockDim.x * gridDim.x;
    }
    __syncthreads();

    int l = blockDim.x;
    while (l >= 1) {
        l /= 2;
        if (threadIdx.x < l) {
            data[threadIdx.x] = (data[threadIdx.x]<data[threadIdx.x + l])? data[threadIdx.x]:data[threadIdx.x + l];
        }
        __syncthreads();
    }
    *min_timestep = data[0];
    // printf("%f\n", min_timestep);


}

__global__
void cudaResampleKernel(
    float* dev_resample_X, 
    float* dev_X, 
    float* dev_accu_time, 
    const int N, 
    const int T) {
    // TODO
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    while (idx < N) {
        int i = (int)(dev_accu_time[idx] * 10);
        // printf("inside resample kernel, idx: %d, i: %d\n", idx, i);
        for (int j = 0; j < i && j < T; ++j) {
            printf("j: %d , T: %d , i: %d , idx: %d , dex_X: %f , dev_resample: %f\n", j, T, i, idx, dev_X[idx], dev_resample_X[idx]);
            if (dev_resample_X[idx * T + j] < 0) {
                dev_resample_X[idx * T + j] = 1.0;
                printf("%d %d, %f\n",idx, j, dev_X[idx]);

            }
        }
        // while (dev_is_resampled[idx * T + i] == 0 && i >= 0 && i < T) {
        //     dev_is_resampled[idx * T + i] = 1;
        //     dev_resample_X[idx * T + i] = dev_X[idx];
        //     i--;
        // }
        idx += blockDim.x * gridDim.x;
    }
}





void cudaCallGillKernel(const int blocks,
    const int threadsPerBlock,
    float* dev_points, 
    float* dev_points_2, 
    float* state,
    float* X, 
    float* dev_timestep,
    float* dev_accu_time,
    const int N) {
    cudaGillKernel<<<blocks, threadsPerBlock>>>(dev_points, dev_points_2, state, X, dev_timestep, dev_accu_time, N);
}

void cudaCallFindMinKernel(const int blocks, 
    const int threadsPerBlock,
    float* dev_accu_time,
    float* dev_min_time,
    const int N) {
    cudaFindMinKernel<<<blocks, threadsPerBlock>>>(dev_accu_time, dev_min_time, N);
}


void cudaCallResampleKernel(const int blocks, 
    const int threadsPerBlock, 
    float* dev_resample_X, 
    float* dev_X, 
    float* dev_accu_time, 
    const int N, 
    const int T) {
    cudaResampleKernel<<<blocks, threadsPerBlock>>>(dev_resample_X, dev_X, dev_accu_time, N, T);
}













